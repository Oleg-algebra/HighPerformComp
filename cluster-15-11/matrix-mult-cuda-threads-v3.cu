#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <experimental/filesystem>
#include <math.h>

using namespace std;
__global__ 
void multMatrixVector(int nPoints,int* rows, int*cols ,double *vals, double *v, double *resVector){
    //TODO: rewrite function

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double eps = 1e-13;
    //printf("block: %d -- thread: %d -- start index: %d\n",blockIdx.x,threadIdx.x,index);
    for(int i =index; i<nPoints;i+=stride){
        double value = vals[i] * v[cols[i]];
	if(value > eps || value < -eps){
	    double oldValue = resVector[rows[i]];
	    int count = 0;		
            do{
              resVector[rows[i]] += value;
	      count++;
              //printf("block: %d -- thread: %d -- attempt: %d -- row: %d\n",blockIdx.x,threadIdx.x,count,rows[i]);
	    }while(resVector[rows[i]] == oldValue);	
        }
        
	if(cols[i] == 0){
            //printf("res[1] = %f\n",resVector[1]);
            //printf("block: %d -- thread: %d -- row index: %d\n",blockIdx.x,threadIdx.x,rows[i]);
            //printf("value written to vector: %f  -- resVector[%d] = %f\n",value,rows[i],resVector[rows[i]]);
	    //printf("col: %d -- resV[%d] = %f -- v[%d] = %f -- value: %f\n",cols[i],rows[i],resVector[rows[i]],cols[i], v[cols[i]],value);
	}
        

    }

}

void getData(const string& dataString, double *storage){
    stringstream ss(dataString);
    string singleData;
    int i = 0;
    while (getline(ss,singleData,' ')) {
//        cout << "Data: "<< singleData << "\n";
        storage[i] = std::stod(singleData);
        i++;
        // store token string in the vector

    }
}

void readMatrix(int *cols,int *rows, double *vals, string& fileName){

    fstream file;
    file.open(fileName,ios::in);
    int counter = 0;
    if(file.is_open()){
        string text;
        double *values = new double[3];
        getline(file,text);
        getData(text,values);
        
        while(getline(file,text)){
            getData(text,values);
            cols[counter] = (int)values[1];
	    rows[counter] = (int)values[0];
	    vals[counter] = values[2];
            counter++;
            }
        file.close();
    } else{
        cout << "file closed"<<"\n";
    }
    cout<<"data written: "<<counter<<"\n";
}

void readHead(const string &fileName, int *headData){
    fstream file;
    file.open(fileName,ios::in);
    double *values = new double[3];
    if(file.is_open()){
        string text;
        
        getline(file,text);
        getData(text,values);
		headData[0] = values[0];
		headData[1] = values[1];
		headData[2] = values[2];
        file.close();
    } else{
        cout << "file closed"<<"\n";
    }
}

void printMatrix(int n, int* rows, int* cols, double* vals){
	for(int i = 0; i<n;i++){
		printf("col: %d -- row: %d -- val: %f\n",rows[i],cols[i],vals[i]);
	}
}

void printVector(int n, double*v){
    printf("printing vector\n");
    for(int i = 0;i<n;i++){
        printf("v[%d] = %f\n",i,v[i]);
    }
}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %d\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %d\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %d\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i){
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    }
    for (int i = 0; i < 3; ++i){
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    }
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %d\n",  devProp.totalConstMem);
    printf("Texture alignment:             %d\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));

}

void start(int numBlocks, int blockSize, string path){
    
    double *v, *res, *vals;
    int *rows, *cols;
    cout<<"memory allocation starting..."<<"\n";
    int *head = new int[3];
    cout<<"reading head\n";
    readHead(path,head);
    cout<<"Head obtained\n";
    int nPoints = head[2];
    int N = head[0];
    cout<<"data points number: "<<(int)nPoints<<"\n";
    hipMallocManaged(&cols, nPoints*sizeof(int));
    hipMallocManaged(&rows, nPoints*sizeof(int));
    hipMallocManaged(&vals, nPoints*sizeof(double));
    hipMallocManaged(&v, N*sizeof(double));
    hipMallocManaged(&res, N*sizeof(double));  
    cout<<"memory allocated"<<"\n";
    cout<<"total size memory: "<<(2*nPoints*sizeof(int)+nPoints*sizeof(double)+2*N*sizeof(double))/1024<<" KB\n";
    
    cout<<"reading data from file\n";
    readMatrix(cols,rows,vals,path);
    cout<<"Data reading Finished\n";

    cout<<"filling v and res with zeros\n";
    for(int i = 0; i<N; i++){
	v[i] = 0.0;
	res[i] = 0.0;
    }
    v[0] = 1.0;
    
    /*
    cout<<"printing matrix\n";
    printMatrix(22,rows,cols,vals);
    cout<<"printing vector v\n";
    printVector(10,v);
    cout<<"printing vector res\n";
    printVector(10,res);
    

    int blockSize = 256;
    int numBlocks = (nPoints + blockSize - 1) / blockSize;
    */

    cout<<"BlockSize: "<<blockSize<<"\n";
    cout<<"numBlocks: "<<numBlocks<<"\n";
    cout<<"numberOfThreads: "<<numBlocks*blockSize<<"\n";

    cout<<"Starting computation\n";
    multMatrixVector<<<numBlocks,blockSize>>>(nPoints,rows,cols,vals,v,res);
    cout<<"Computation finished\n";
    hipDeviceSynchronize();
    //printVector(10,res);
    double maxError = 0.0;
    //printVector(10,res);

    int ind = 0;
    for (int i = 0; i < N; i++){
        if(cols[ind] == 0 && rows[ind] == i){	
            maxError = fmax(maxError, fabs(res[i]-vals[ind]));
            ind++;
        }
    }
    
    cout << "Max error: " << maxError << "\n";
    /*
    double eps = 1e-10;
    for(int i = 0; i<N;i++){
        if(fabs(res[i])>eps){
            printf("res[%d] = %f\n",i,res[i]);
        }
    }*/
    
    if(true){
        fstream outFile;
        outFile.open("log_file.txt",ios::app);
        outFile<<"blockSize: "<<blockSize<<"\n";
        outFile<<"numBlocks: "<<numBlocks<<"\n";
	outFile<<"numberOfThreads: "<<numBlocks*blockSize<<"\n";
	outFile<<"pointsNumber: "<<nPoints<<"\n";
        outFile<<"Max error: "<< maxError <<"\n";
        outFile<<"============================\n";
        outFile.close();   
    }

    if(true){
        fstream resFile;
        resFile.open("corr_res.txt",ios::out);
        resFile<<N<<"\n";
        for(int i = 0; i<N; i++){
	    resFile<<res[i]<<"\n";
	}
        resFile.close();   
    }

    
    delete [] head;
    hipFree(v);
    hipFree(res);
    hipFree(vals);
    hipFree(rows);
    hipFree(cols);
    hipDeviceSynchronize();
}

void printGPUInfo(){
    int deviceCount = 0;
    hipError_t err = hipSuccess;
    err = hipGetDeviceCount(&deviceCount);
    if(err == hipSuccess){
         cout<<"deviceCount: "<<deviceCount<<"\n";
    
   
         for(int id = 0; id < deviceCount;id++){
         size_t totalDevMem, freeDevMem;
         err = hipSetDevice(id);
         if (err == hipSuccess) {
                     
                  hipMemGetInfo(&freeDevMem, &totalDevMem);
                  cout << " : ";
         	  cout << "Dev " << id << " (" << (freeDevMem/1024) << " KB of " << (totalDevMem/1048576) << " MB free)\n";

             }
        }
    }
}


int main() {
    string  path;
    //path = "matrices/sparsine/sparsine2.mtx";
    //path = "matrices/newSparsine2.txt";
    //path = "matrices/test-matrix.txt";
    path = "matrices/sparsine.mtx";

    int *head = new int[3];
    cout<<"reading head\n";
    readHead(path,head);
    cout<<"Head obtained\n";
    int nPoints = head[2];
    
    //printGPUInfo();    

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, 0);
    //printDevProp(prop);
    cout << "Multiprocessor Count: " << prop.multiProcessorCount << endl;
    cout << "Thread Count: " << prop.maxThreadsDim[2] << endl;

    for(int bs = 32; bs<=32; bs+=32){	
        int blockSize = prop.maxThreadsDim[0];
        int numBlocks = min(prop.multiProcessorCount, (nPoints + blockSize - 1)/ blockSize);
    
        start(numBlocks,blockSize,path);
    }
    delete [] head;
    return 0;
}
