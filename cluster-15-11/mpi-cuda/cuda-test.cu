/* multiply.cu */

#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ 
void multiply(int N, float *a, float *b){
    a[0] = 1.0f;
    b[0]= 1.0f;
}

extern "C" void launch_multiply(int N, float *a, float *b)
{
    float *a_gpu, *b_gpu;
    hipMallocManaged(&a_gpu, N*sizeof(float));
    hipMallocManaged(&b_gpu, N*sizeof(float));
	
    for(int i = 0; i<N; i++){
		a_gpu[i] = a[i];
		b_gpu[i] = b[i];
    }
	
    printf("cuad-test: N = %d\n",N);
    multiply<<< 1 , 1 >>> (N,a_gpu, b_gpu);
    hipDeviceSynchronize();
    
    for(int i = 0; i<N; i++){
		a[i] = a_gpu[i];
		b[i] = b_gpu[i];
	}
	hipFree(a_gpu);
	hipFree(b_gpu);
}