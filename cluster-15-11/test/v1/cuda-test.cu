/* multiply.cu */

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <iostream>
#include <fstream>
#include <experimental/filesystem>

using namespace std;


__global__ 
void multMatrixVector(int nPoints,int* rows, int*cols ,double *vals, double *v, double *resVector){
    //TODO: rewrite function

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double eps = 1e-13;
    //printf("block: %d -- thread: %d -- start index: %d\n",blockIdx.x,threadIdx.x,index);
    for(int i =index; i<nPoints;i+=stride){
        double value = vals[i] * v[cols[i]];
	if(value > eps || value < -eps){
	    double oldValue = resVector[rows[i]];
	    int count = 0;		
            do{
              resVector[rows[i]] += value;
	      count++;
              //printf("block: %d -- thread: %d -- attempt: %d -- row: %d\n",blockIdx.x,threadIdx.x,count,rows[i]);
	    }while(resVector[rows[i]] == oldValue);	
        }
	
	if(cols[i] == 0){
            //printf("res[16668] = %f\n",resVector[16668]);
            //printf("block: %d -- thread: %d -- row index: %d\n",blockIdx.x,threadIdx.x,rows[i]);
            //printf("value written to vector: %f  -- resVector[%d] = %f\n",value,rows[i],resVector[rows[i]]);
	    //printf("col: %d -- resV[%d] = %f -- v[%d] = %f -- value: %f\n",cols[i],rows[i],resVector[rows[i]],cols[i], v[cols[i]],newValue);
	                	
	}

    }

}

void getData(const string& dataString, double *storage){
    stringstream ss(dataString);
    string singleData;
    int i = 0;
    while (getline(ss,singleData,' ')) {
//        cout << "Data: "<< singleData << "\n";
        storage[i] = std::stod(singleData);
        i++;
        // store token string in the vector

    }
}

void readMatrix(int *cols,int *rows, double *vals, string& fileName){

    fstream file;
    file.open(fileName,ios::in);
    int counter = 0;
    if(file.is_open()){
        string text;
        double *values = new double[3];
        getline(file,text);
        getData(text,values);
        
        while(getline(file,text)){
            getData(text,values);
            cols[counter] = (int)values[1];
	    rows[counter] = (int)values[0];
	    vals[counter] = values[2];
            counter++;
            }
        file.close();
    } else{
        cout << "file closed"<<"\n";
    }
    
}

void readHead(const string &fileName, int *headData){
    fstream file;
    file.open(fileName,ios::in);
    double *values = new double[3];
    if(file.is_open()){
        string text;
        
        getline(file,text);
        getData(text,values);
		headData[0] = values[0];
		headData[1] = values[1];
		headData[2] = values[2];
        file.close();
    } else{
        cout << "file closed"<<"\n";
    }
}

void printMatrix(int n, int* rows, int* cols, double* vals){
	for(int i = 0; i<n;i++){
		printf("col: %d -- row: %d -- val: %f\n",rows[i],cols[i],vals[i]);
	}
}

void printVector(int n, double*v){
    printf("printing vector\n");
    for(int i = 0;i<n;i++){
        printf("v[%d] = %f\n",i,v[i]);
    }
}


extern "C" void launch_multiply(int rank, double *vector, double *resVector)
{
    //printf("rank %d choosing GPU....\n",rank);
    hipSetDevice(rank);
    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, 0);

    double *vector_gpu, *resVector_gpu, *vals, *validationVec;
    int *cols, *rows;
    
    int *head = new int[3];  
    string fileName = "chunk_" + std::to_string(rank)+".txt";
    readHead(fileName, head);
    int N = head[0];
    int nPoints = head[2];
    //printf("rank: %d -- N=%d\n",rank,N);
    
    printf("rank %d allocating memory...\n",rank);
    hipMallocManaged(&vector_gpu, N*sizeof(double));
    hipMallocManaged(&resVector_gpu, N*sizeof(double));
    hipMallocManaged(&validationVec, N*sizeof(double));

    hipMallocManaged(&vals, nPoints*sizeof(double));
    hipMallocManaged(&rows, nPoints*sizeof(int));
    hipMallocManaged(&cols, nPoints*sizeof(int));
    printf("rank %d memory allocation finished...\n",rank);
    
    printf("rank %d reading chunk_%d\n",rank,rank);
    readMatrix(cols,rows,vals,fileName);
    printf("rank %d data written %d\n",rank,nPoints);
    printf("rank %d finished reading chunk_%d\n",rank,rank);
    /*
    for(int i = 0; i<5; i++){
	printf("rank: %d -- row: %d -- col: %d  -- val: %f\n",rank,rows[i],cols[i],vals[i]);
    }*/
    	
    printf("rank %d copy data to GPU.....\n",rank);
    for(int i = 0; i<N; i++){
		vector_gpu[i] = vector[i];
		resVector_gpu[i] = 0.0;
                validationVec[i] = 0.0;
    }
    //printf("rank %d vec[0] = %f\n",rank,vector_gpu[0]);


    int blockSize = prop.maxThreadsDim[2];
    int numBlocks = min(prop.multiProcessorCount, (nPoints + blockSize - 1)/ blockSize);	
    //cout<<"blockSize: "<<blockSize<<"\n";
    //cout<<"numBlocks: "<<numBlocks<<"\n";
    printf("rank %d starting computations\n",rank);	
    multMatrixVector<<< blockSize,numBlocks >>> (nPoints,rows,cols,vals,vector_gpu, resVector_gpu);
    multMatrixVector<<< 1,1 >>>(nPoints,rows,cols,vals,vector_gpu, validationVec);
    printf("rank %d finished computations\n",rank);
    hipDeviceSynchronize();
    printf("rank %d copy data to CPU.....\n",rank);
    //printf("rank %d res[%d] = %f\n",rank,rank, resVector_gpu[rank]);
    for(int i = 0; i<N; i++){
		vector[i] = vector_gpu[i];
		resVector[i] = resVector_gpu[i];
    }

    double maxError = 0.0;
    int ind = 0;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(resVector_gpu[i]-validationVec[i]));
    }
    
    printf("rank %d -- Max error = %f\n",rank,maxError);

    printf("rank %d free GPU memory\n",rank);
    hipFree(validationVec);
    hipFree(vals);
    hipFree(rows);
    hipFree(cols);
    hipFree(vector_gpu);
    hipFree(resVector_gpu);
}